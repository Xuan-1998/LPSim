#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    
    int ngpus;
    hipGetDeviceCount(&ngpus);
    printf("Number of gpus: %d\n",ngpus);
    size_t size = 1 << 20;
    float *d_data_src, *d_data_dst;
    int gpuid1 = 0; 
    int gpuid2 = 1; 
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float *h_data = new float[size];
    for(size_t i = 0; i < size; i++) {
        h_data[i] = 1.0f;
    }
    
    hipSetDevice(gpuid1);
    hipMalloc((void **)&d_data_src, size * sizeof(float));

    hipEventRecord(start);
    hipMemcpy(d_data_src, h_data, size * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time hipMemcpyHostToDevice: " << milliseconds << " ms\n";



    hipSetDevice(gpuid2);
    hipMalloc((void **)&d_data_dst, size * sizeof(float));
    int canAccessPeer = 0;
    hipDeviceCanAccessPeer(&canAccessPeer, gpuid1, gpuid2);
    if (canAccessPeer) {
        hipDeviceEnablePeerAccess(gpuid2, 0);
    }
    else{
        printf("Cannot Enable PeerAccess\n");
    }


    //peer to peer
    hipSetDevice(gpuid1);
    hipEventRecord(start);

    hipMemcpyPeer(d_data_dst, gpuid2, d_data_src, gpuid1, size * sizeof(float));

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time communication: " << milliseconds << " ms\n";

    hipFree(d_data_src);
    hipFree(d_data_dst);
    delete[] h_data;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //----hipMallocManaged-------
    {
    float *d_data;
    hipMallocManaged(&d_data, size * sizeof(float));
    // memcpy(d_data, h_data, size * sizeof(float));
    hipMemset(d_data, 2.5, size * sizeof(float));

    // to gpu1
    hipSetDevice(gpuid1);
    hipMemPrefetchAsync(d_data, size * sizeof(float), gpuid1);
    hipDeviceSynchronize();

    //to gpu2
    hipSetDevice(gpuid2);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMemPrefetchAsync(d_data, size * sizeof(float), gpuid2);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time Unified memory: " << milliseconds << " ms" << std::endl;

    hipFree(d_data);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    }
    

    return 0;
}
