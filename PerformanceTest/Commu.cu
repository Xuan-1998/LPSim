#include <hip/hip_runtime.h>
#include <iostream>
#include<vector>
#include <fstream>
int main() {
    
    int ngpus;
    float *d_data_src, *d_data_dst;
    int gpuid1 = 0; 
    int gpuid2 = 1; 
    hipGetDeviceCount(&ngpus);
    printf("Number of gpus: %d\n",ngpus);
    int canAccessPeer = 0;
    hipDeviceCanAccessPeer(&canAccessPeer, gpuid1, gpuid2);
    if (canAccessPeer) {
        hipDeviceEnablePeerAccess(gpuid2, 0);
        printf("Enable PeerAccess\n");
    }
    else{
        printf("Cannot Enable PeerAccess\n");
    }
    // size_t size = (1 << 20)*10;
   
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    
    std::vector<size_t> sizes = {
    1UL << 10, 10UL << 10, 100UL << 10,   // 1KB, 10KB, 100KB
    1UL << 20, 10UL << 20, 100UL << 20,   // 1MB, 10MB, 100MB
    1UL << 30, 2UL << 30   // 1GB, 2GB, 4GB
    };
    for (auto& size : sizes) {
        size /= sizeof(float);
    }
     std::ofstream csvFile("timing_commu.csv");
     csvFile << "Size,Transfer Type,Time (ms)\n";

    if (!csvFile.is_open()) {
        std::cerr << "Unable to open file for writing.\n";
        return -1;
    }
    for (auto size : sizes){
    float *h_data = new float[size];
    for(size_t i = 0; i < size; i++) {
        h_data[i] = 1.0f;
    }
    
   //prepare data
    hipSetDevice(gpuid1);
    hipMalloc((void **)&d_data_src, size * sizeof(float));
    hipEventRecord(start);
    hipMemcpy(d_data_src, h_data, size * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time hipMemcpyHostToDevice: " << milliseconds << " ms\n";
    

    hipSetDevice(gpuid2);
    hipMalloc((void **)&d_data_dst, size * sizeof(float));
    
    
    //device-host-device
    float* h_data_tmp = new float[size];
    hipSetDevice(gpuid1);
    hipEventRecord(start);
    hipMemcpy(h_data_tmp, d_data_src, size* sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_data_src, h_data_tmp, size * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time D-H-D: " << milliseconds << " ms\n";
    csvFile << size * sizeof(float) << ",D-H-D," << milliseconds << "\n";

    //peer to peer
    hipSetDevice(gpuid1);
    hipEventRecord(start);
    hipMemcpyPeer(d_data_dst, gpuid2, d_data_src, gpuid1, size * sizeof(float));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time P2P: " << milliseconds << " ms\n";
    csvFile << size * sizeof(float) << ",P2P," << milliseconds << "\n";

    hipFree(d_data_src);
    hipFree(d_data_dst);
    delete[] h_data;

    //----hipMallocManaged-------
    
    float *d_data;
    hipMallocManaged(&d_data, size * sizeof(float));
    // memcpy(d_data, h_data, size * sizeof(float));
    hipMemset(d_data, 2.5, size * sizeof(float));

    // to gpu1
    hipSetDevice(gpuid1);
    hipMemPrefetchAsync(d_data, size * sizeof(float), gpuid1);
    hipDeviceSynchronize();

    //to gpu2
    hipSetDevice(gpuid2);
    hipEventRecord(start);
    hipMemPrefetchAsync(d_data, size * sizeof(float), gpuid2);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time Unified memory: " << milliseconds << " ms" << std::endl;
    hipFree(d_data);
}
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
