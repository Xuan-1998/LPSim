#include <hip/hip_runtime.h>
#include <iostream>

__global__ void readAndWriteKernel(float *data, float *buffer, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        //simulate read and write
        buffer[idx]=data[idx];
        data[idx] = 2*buffer[idx];
    }
}
__global__ void readKernel(float *data, float *output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = data[idx];
    }
}

__global__ void writeKernel(float *data, float value, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = value;
    }
}
int main() {
    size_t size = 1 << 20; // 1M byte
    float *d_data, *d_output;
    float value = 2.0f;

    hipMalloc((void **)&d_data, size * sizeof(float));
    hipMalloc((void **)&d_output, size * sizeof(float));

    float *h_data = new float[size];
    for(size_t i = 0; i < size; i++) {
        h_data[i] = 1.0f; 
    }
    hipMemcpy(d_data, h_data, size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((size + blockSize.x - 1) / blockSize.x);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // read
    hipEventRecord(start);
    readKernel<<<gridSize, blockSize>>>(d_data, d_output, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time read: " << milliseconds << " ms\n";

    

    // write
    hipEventRecord(start);
    writeKernel<<<gridSize, blockSize>>>(d_data, value, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time write: " << milliseconds << " ms\n";


    // read and write
    hipEventRecord(start);
    readAndWriteKernel<<<gridSize, blockSize>>>(d_data,d_output, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time read and write: " << milliseconds << " ms\n";


    hipFree(d_data);
    hipFree(d_output);
    delete[] h_data;

    return 0;
}
