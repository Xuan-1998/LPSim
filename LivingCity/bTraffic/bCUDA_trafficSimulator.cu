#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------------------------------------------------
// Copyright 2017, 2018 Purdue University, Ignacio Garcia Dorado, Daniel Aliaga
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that the 
// following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the 
// following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the 
// following disclaimer in the documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its contributors may be used to endorse or promote 
// products derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS AS IS AND ANY EXPRESS OR IMPLIED WARRANTIES, 
// INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE 
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR 
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, 
// WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE 
// USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//---------------------------------------------------------------------------------------------------------------------

//CUDA CODE
#include <stdio.h>
#include <vector>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""

#include "bTrafficPeople.h"
#include "bEdgeIntersectionData.h"

//#include "BaseTsd.h"//for U64
#include "bTrafficConstants.h"

#ifndef ushort
#define ushort uint16_t
#endif
#ifndef uint
#define uint uint32_t
#endif
#ifndef uchar
#define uchar uint8_t
#endif

#define DEBUG_TRAFFIC 0

///////////////////////////////
// CONSTANTS (also change in bTrafficConstants)
__constant__ float deltaTime = 0.5f;
__constant__ float cellSize = 1.0f;//
__constant__ float vCT2MS = 1.0f / 0.5f;//const float vCT2MS = cellSize / deltaTime;


// Kernel that executes on the CUDA device
__global__ void square_array()//(float *a, int N)
{
  //int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //if (idx<N) a[idx] = a[idx] * a[idx];
  printf("a\n");
}

void bExampleCUDA(void) {

  float *a_d;//*a_h  // Pointer to host & device arrays
  const int N = 1;  // Number of elements in arrays
  size_t size = N * sizeof(float);
  //a_h = (float *)malloc(size);        // Allocate array on host
  hipMalloc((void **) &a_d, size);   // Allocate array on device
  // Initialize host array and copy it to CUDA device
  //for (int i=0; i<N; i++) a_h[i] = (float)i;
  //hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  // Do calculation on device:
  //int block_size = 1;
  //int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  //square_array <<< n_blocks, block_size >>> (a_d, N);
  square_array << < 1, 1 >> > ();
  // Retrieve result from device and store it in host array
  //hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
  // Print results
  //for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  // Cleanup
  //free(a_h); 
  hipFree(a_d);
}//


//__constant__ float deltaTimeG=1.0f*(1.0f/3600.0f);
//__constant__ float s_0=7.0f;
//const float deltaTimeC=1.0f*(1.0f/3600.0f);
////////////////////////////////
// VARIABLES
/*LC::CUDATrafficPerson *trafficPersonVec_d;
//ushort *nextEdgeM_d;

float cellSize;
float deltaTime;





LC::intersectionData *intersections_d;
uchar *trafficLights_d;*/

bool readFirstMap = true;
//uint mapToReadShift;
//uint mapToWriteShift;
LC::SimulationSt simulationSt;
ushort maxWidthL;

LC::BEdgesDataCUDA edgesData;
LC::BEdgesDataCUDA* edgesData_d;


unsigned long *laneMap_d[2];
uint laneMapSizeL;
BTrafficPeopleCUDA pC;
BTrafficPeopleCUDA* pC_d;

// VARIABLES

template <typename T>
void allocateAndCopy(std::vector<T>& vec, void** pointer) {
  if (vec.size() <= 0) {
    printf("ERROR: allocateAndCopy with size 0\n");
    return;
  }
  hipError_t err;
  size_t size = vec.size()*sizeof(vec[0]);
  err = hipMalloc(pointer, size);   // Allocate array on device
  if (hipSuccess != err)fprintf(stderr, "ac0 Cuda error: %s.\n", hipGetErrorString(err));
  err = hipMemcpy(*pointer, vec.data(), size, hipMemcpyHostToDevice);
  if (hipSuccess != err)fprintf(stderr, "ac1 Cuda error: %s.\n", hipGetErrorString(err));
}

void bInitCUDA(ushort _maxWidthL, LC::BTrafficPeople& people, LC::BEdgesData& _edgesData, std::vector<unsigned long>(&laneMapL)[2]) {//, std::vector<LC::intersectionData>& intersections){
  printf("bInitCUDA\n");
  maxWidthL = _maxWidthL;
  hipError_t err;
  // people
  /*size_t size = people.numPeople;
  err = hipMalloc((void **)&pC.init_intersection, size*sizeof(unsigned short));   // Allocate array on device
  if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
  err = hipMemcpy(pC.init_intersection, people.init_intersection.data(), size, hipMemcpyHostToDevice);
  if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );*/
  allocateAndCopy(people.init_intersection, (void **) &pC.init_intersection);
  allocateAndCopy(people.end_intersection, (void **) &pC.end_intersection);
  allocateAndCopy(people.time_departure, (void **) &pC.time_departure);
  allocateAndCopy(people.active, (void **) &pC.active);
  allocateAndCopy(people.v, (void **) &pC.v);
  allocateAndCopy(people.a, (void **) &pC.a);
  allocateAndCopy(people.b, (void **) &pC.b);
  allocateAndCopy(people.T, (void **) &pC.T);
  allocateAndCopy(people.carType, (void **) &pC.carType);
  ///nextEdge
  allocateAndCopy(people.indTo1stEdge, (void **) &pC.indTo1stEdge);
  allocateAndCopy(people.nextEdge, (void **) &pC.nextEdge);
  //simulation
  allocateAndCopy(people.currIndEdge, (void **) &pC.currIndEdge);
  allocateAndCopy(people.posInLaneC, (void **) &pC.posInLaneC);
  allocateAndCopy(people.laneInEdge, (void **) &pC.laneInEdge);
  // structure itself
  size_t sizeS = sizeof(BTrafficPeopleCUDA);
  err = hipMalloc((void **) &pC_d, sizeS);   // Allocate array on device
  if (hipSuccess != err)fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
  err = hipMemcpy(pC_d, &pC, sizeS, hipMemcpyHostToDevice);
  if (hipSuccess != err)fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));

  //edgeData
  allocateAndCopy(_edgesData.numLinesB, (void **) &edgesData.numLinesB);
  allocateAndCopy(_edgesData.nextInters, (void **) &edgesData.nextInters);
  allocateAndCopy(_edgesData.lengthC, (void **) &edgesData.lengthC);
  allocateAndCopy(_edgesData.maxSpeedCpSec, (void **) &edgesData.maxSpeedCpSec);
  // structure itself
  size_t sizeE = sizeof(LC::BEdgesDataCUDA);
  err = hipMalloc((void **) &edgesData_d, sizeE);   // Allocate array on device
  if (hipSuccess != err)fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
  err = hipMemcpy(edgesData_d, &edgesData, sizeE, hipMemcpyHostToDevice);
  if (hipSuccess != err)fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
  // structure itfelf
  /*size_t sizeD = edgesData.size() * sizeof(LC::BEdgeData);
  err = hipMalloc((void **)&edgesData_d, sizeD);   // Allocate array on device
  if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
  err=hipMemcpy(edgesData_d,edgesData.data(),sizeD,hipMemcpyHostToDevice);
  if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );*/


  //laneMap
  size_t sizeL = laneMapL[0].size() * sizeof(unsigned long);
  err = hipMalloc((void **) &laneMap_d[0], sizeL);   // Allocate array on device
  if (hipSuccess != err)fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
  //err=hipMemcpy(laneMap_d,laneMapL.data(),sizeL,hipMemcpyHostToDevice);
  err = hipMemset(laneMap_d[0], -1, sizeL);//init to -1
  if (hipSuccess != err)fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
  err = hipMalloc((void **) &laneMap_d[1], sizeL);   // Allocate array on device
  if (hipSuccess != err)fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
  //err=hipMemcpy(laneMap_d,laneMapL.data(),sizeL,hipMemcpyHostToDevice);
  err = hipMemset(laneMap_d[1], -1, sizeL);//init to -1
  if (hipSuccess != err)fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
  laneMapSizeL = laneMapL[0].size();
  /*// intersections
  size_t sizeI = intersections.size() * sizeof(LC::intersectionData);
  err=hipMalloc((void **) &intersections_d, sizeI);   // Allocate array on device
  if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
  err=hipMemcpy(intersections_d,intersections.data(),sizeI,hipMemcpyHostToDevice);
  if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
  size_t sizeT = (halfLaneMap/_maxWidthL) * sizeof(uchar);//total number of lanes
  err=hipMalloc((void **) &trafficLights_d, sizeT);   // Allocate array on device
  if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );*/
}//

void bFinishCUDA(void) {
  //////////////////////////////
  // FINISH
  printf("bFinishCUDA\n");
  hipFree(pC.init_intersection);
  hipFree(pC.end_intersection);
  hipFree(pC.time_departure);
  hipFree(pC.active);
  hipFree(pC.v);
  hipFree(pC.a);
  hipFree(pC.b);
  hipFree(pC.init_intersection);
  hipFree(pC.T);
  hipFree(pC.carType);
  hipFree(pC.indTo1stEdge);
  hipFree(pC.nextEdge);

  hipFree(pC.currIndEdge);
  hipFree(pC.posInLaneC);
  hipFree(pC.laneInEdge);

  //hipFree(nextEdgeM_d);
  hipFree(edgesData_d);
  hipFree(laneMap_d);
  //hipFree(intersections_d);
  //hipFree(trafficLights_d);

}//

void bGetDataCUDA(LC::BTrafficPeople& people) {//,std::vector<uchar>& trafficLights){
  // printf("bGetDataCUDA\n");
  // copy back people
  //
  //hipMemcpy(trafficPersonVec.data(),trafficPersonVec_d,size,hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
  size_t sizeA = people.active.size() * sizeof(unsigned char);
  hipMemcpy(people.active.data(), pC.active, sizeA, hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
  size_t size = people.currIndEdge.size() * sizeof(unsigned int);
  hipMemcpy(people.currIndEdge.data(), pC.currIndEdge, size, hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
  size_t sizeP = people.posInLaneC.size() * sizeof(float);
  hipMemcpy(people.posInLaneC.data(), pC.posInLaneC, sizeP, hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
  size_t sizeL = people.laneInEdge.size() * sizeof(unsigned char);
  hipMemcpy(people.laneInEdge.data(), pC.laneInEdge, sizeL, hipMemcpyDeviceToHost);//hipMemcpyHostToDevice

  //size_t sizeI = trafficLights.size() * sizeof(uchar);
  //hipMemcpy(trafficLights.data(),trafficLights_d,sizeI,hipMemcpyDeviceToHost);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__
unsigned short usmax(unsigned short a, unsigned b) {
  return (a < b) ? b : a;
}//
__global__
void kernel_trafficSimulation(
//uint p,
uint numPeople,
LC::SimulationSt simulationSt,
const ushort maxWidthL,
BTrafficPeopleCUDA& people,
LC::BEdgesDataCUDA& edgesData,
unsigned long* laneMapR,//,
unsigned long* laneMapW
//std::vector<intersectionData>& intersections,
//std::vector<uchar>& trafficLights
) {

  int p = blockIdx.x * blockDim.x + threadIdx.x;
  //printf("C p %d Numpe %d\n",p,numPeople);
  if (p >= numPeople) {//CUDA check (outside margins)
    return;
  }
  //printf("currentTime %f   0 Person: %d State %d\n", simulationSt.currentTime, p, people.active[p]);
  if (DEBUG_TRAFFIC == 1)
    printf("currentTime %f   0 Person: %d State %d Time Dep %f\n", simulationSt.currentTime, p, people.active[p], people.time_departure[p]);
  // return;
  uchar active = people.active[p];
  uchar* laneMapWC = (uchar*) laneMapW;
  uchar* laneMapRC = (uchar*) laneMapR;
  ///////////////////////////////
  //0. check if finished
  if (active == 2) {
    return;
  }
  ///////////////////////////////
  //1. check if person should still wait or should start
  if (active == 0) {

    //printf("  1. Person: %d active==0\n",p);
    if (people.time_departure[p] > simulationSt.currentTime) {//wait
      //1.1 just continue waiting
      //printf("   1.1 Person: %d wait\n",p);
      return;
    } else {//start
      //1.2 find first edge
      people.currIndEdge[p] = people.indTo1stEdge[p];//pointer to 1st edge in nextEdge
      ushort firstEdge = people.nextEdge[people.currIndEdge[p]];
      ushort numOfCells = edgesData.lengthC[firstEdge];// ceil(trafficPersonVec[p].length / cellSize);
      if (firstEdge == 0xFFFF || numOfCells < 8) {//already in destination or not edge
        people.active[p] = 2;
        return;
      }
      if (DEBUG_TRAFFIC == 1)printf("   1.2 Person: %d TRY put in first edge\n", p, firstEdge);

      //1.4 try to place it in middle of edge

      //ushort numOfCellsL = ((numOfCells +7) & ~7u)/8;//round up
      ushort numOfCellsL = ((numOfCells) / 8);//not try last
      ushort initShiftL = (ushort) numOfCellsL*0.25f; //(~half of road)

      //bool placed = false;
      unsigned long laneL;
      ushort b;
      ushort lN = 0;// edgesData.numLinesB[firstEdge] - 1;//just right LANE !!!!!!!
      if (DEBUG_TRAFFIC == 1) printf("   1.2 Person: %d TRY put in first edge--> %u numOfCellsL %u n %u\n", p, firstEdge, numOfCellsL, numOfCells);
      for (ushort B = initShiftL; B < numOfCellsL; B++) {

        laneL = laneMapR[maxWidthL*(firstEdge + lN) + B];//get byte of edge (proper line)

        b = usmax(//std::max<ushort>(
          (laneL & 0x00000000FFFFFFFF == 0x00000000FFFFFFFF) * 1,
          (laneL & 0xFFFFFFFF00000000 == 0xFFFFFFFF00000000) * 4);

        if (b == 0)
          continue;
        people.posInLaneC[p] = b + B * 8;//cells
        laneMapWC[(maxWidthL*(firstEdge + lN) + B) * 8 + b] = 0;// (uchar)(trafficPersonVec[p].v * 3);//speed in m/s *3 (to keep more precision
        laneMapRC[(maxWidthL*(firstEdge + lN) + B) * 8 + b] = 0;// also in read to avoid put two in the same place
        people.v[p] = 0;
        people.active[p] = 1;
        people.laneInEdge[p] = lN;
        return;//placed
      }
      return;//no posible to place now
    }
  }
  ///////////////////////////////
  //2. it is moving
  ushort cEdge = people.nextEdge[people.currIndEdge[p]];//curent edge
  ushort cEdgeLengthC = edgesData.lengthC[cEdge];
  float v = people.v[p];//velocity
  uchar lN = people.laneInEdge[p];//lane
  float posInLaneC = people.posInLaneC[p];//position in lane

  float dv_dt = people.a[p] * (1.0f - std::pow((v / edgesData.maxSpeedCpSec[cEdge]), 4.0f));
  float numCToMove = fmax(0.0f, v*deltaTime + 0.5f*(dv_dt) *deltaTime*deltaTime);

  v = fmax(v + dv_dt*deltaTime, 0.0f);
  if (DEBUG_TRAFFIC == 1)
    //if (p%1000==0)
    printf("p[%d] v %f--> v %f || dv_dt %f numCToMove %f maxSpeed %f\n", p, people.v[p], v, dv_dt, numCToMove, edgesData.maxSpeedCpSec[cEdge]);
  people.v[p] = v;

  //2.1 was not really moving
  if (v == 0) {
    laneMapWC[(maxWidthL*(cEdge + lN)) * 8 + (uint)(posInLaneC)] = 0;
    return;
  }

  //2.2 move
  bool goNextEdge = false;
  posInLaneC += numCToMove;//new possition
  if (posInLaneC > cEdgeLengthC) {//reach intersection
    posInLaneC -= cEdgeLengthC;
    goNextEdge = true;
  }

  // move next edge (when it is not bicycle)
  if (goNextEdge == true && people.carType[p] != 1) {

    people.currIndEdge[p]++;//move next edge
    cEdge = people.nextEdge[people.currIndEdge[p]];//curent edge
    if (DEBUG_TRAFFIC == 1)printf("    2.2 Person: %d new edge %u\n", p, cEdge);
    if (cEdge == 0xFFFF) {
      if (DEBUG_TRAFFIC == 1)printf("    2.2 Person: %d FINISHED\n", p);
      people.active[p] = 2;//finished
      //onePersonFinished(simulationSt);
      return;
    }

  }
  //Assuming that when cartype =1, it is a bicycle. and bicycle does not need to move to another edge(?)
  if(people.carType[p] == 1){
    laneMapWC[(maxWidthL*(cEdge + lN)) * 8 + uint(posInLaneC)] = v  * vCT2MS;//(assuming that it is not as fast as the normal car?)
  }
  else{
  laneMapWC[(maxWidthL*(cEdge + lN)) * 8 + uint(posInLaneC)] = v * 3 * vCT2MS;//to ms triple
  }
  
  people.posInLaneC[p] = posInLaneC;//position in lane (we assume that it is long enought to hold it)
}



void bSimulateTrafficCUDA(float currentTime, uint numPeople) {//, uint numIntersections){
  //printf("bSimulateTrafficCUDA\n");
  ////////////////////////////////////////////////////////////
  // 1. CHANGE MAP: set map to use and clean the other
  hipError_t err;
  /*if (readFirstMap == true){
          simulationSt.mapToReadShiftL = 0;
          simulationSt.mapToWriteShiftL = halfLaneMapL;
          err = hipMemset(&laneMap_d[halfLaneMapL], -1, halfLaneMapL*sizeof(unsigned long));//clean second half
          if ( hipSuccess != err )fprintf( stderr, "Cuda error M0: %s.\n",hipGetErrorString( err) );
          }else{
          simulationSt.mapToReadShiftL = halfLaneMapL;
          simulationSt.mapToWriteShiftL = 0;
          err = hipMemset(&laneMap_d[0], -1, halfLaneMapL*sizeof(unsigned long));//clean first half
          if (hipSuccess != err)fprintf(stderr, "Cuda error M1: %s.\n", hipGetErrorString(err));
          }*/
  simulationSt.currentTime = currentTime;
  simulationSt.cArray = readFirstMap;//read
  err = hipMemset(&laneMap_d[!readFirstMap][0], -1, laneMapSizeL*sizeof(unsigned long));//clean write
  if (hipSuccess != err)fprintf(stderr, "Cuda error M1: %s.\n", hipGetErrorString(err));

  readFirstMap = !readFirstMap;//next iteration invert use


  //kernel_intersectionSimulation << < numIntersections, 1 >> > (numIntersections, currentTime, intersections_d, trafficLights_d);
  // kernel_trafficSimulation << < ceil(numPeople / 1024.0f), 1024 >> > (numPeople, simulationSt, maxWidthL, pC, edgesData_d, laneMap_d);// , /*intersections_d, trafficLights_d, */mapToReadShift, mapToWriteShift, maxWidthL);
  //printf("K>> numPeople %d currentTime %f\n", numPeople, currentTime/3600.0f);
  kernel_trafficSimulation << < ceil(numPeople / 1024.0f), 1024 >> > (numPeople, simulationSt, maxWidthL, pC, edgesData, laneMap_d[simulationSt.cArray], laneMap_d[!simulationSt.cArray]);// , /*intersections_d, trafficLights_d, */mapToReadShift, mapToWriteShift, maxWidthL);
  err = hipGetLastError();
  if (hipSuccess != err) printf("ERROR: kernel_trafficSimulation: %s.\n", hipGetErrorString(err));
  hipDeviceSynchronize();
  //printf("K<< \n");

}//


/*__device__ void calculateGapsLC(
               float cellSize,
               ushort maxWidth,
               uint mapToReadShift,
               uchar *laneMap,
               uchar trafficLightState,
               ushort laneToCheck,
               float posInMToCheck,
               float length,
               uchar &v_a,
               uchar &v_b,
               float &gap_a,
               float &gap_b){
               ushort numOfCells=ceil(length/cellSize);
               ushort initShift=ceil(posInMToCheck/cellSize);
               uchar laneChar;
               bool found=false;
               // CHECK FORWARD
               //printf("initShift %u numOfCells %u\n",initShift,numOfCells);
               for(ushort b=initShift-1;(b<numOfCells)&&(found==false);b++){//NOTE -1 to make sure there is none in at the same level
               laneChar=laneMap[mapToReadShift+maxWidth*(laneToCheck)+b];
               if(laneChar!=0xFF){
               gap_a=((float)b-initShift)*cellSize;//m
               v_a=laneChar;//laneChar is in 3*ms (to save space in array)
               found=true;
               break;
               }
               }
               if(found==false){
               if(trafficLightState==0x00){//red
               //gap_a=((float)numOfCells-initShift)*cellSize;
               //found=true;
               gap_a=gap_b=1000.0f;//force to change to the line without vehicle
               v_a=v_b=0xFF;
               return;
               }
               }
               if(found==false){
               gap_a=1000.0f;
               }
               // CHECK BACKWARDS
               found=false;
               //printf("2initShift %u numOfCells %u\n",initShift,numOfCells);
               for(int b=initShift+1;(b>=0)&&(found==false);b--){//NOTE +1 to make sure there is none in at the same level
               laneChar=laneMap[mapToReadShift+maxWidth*(laneToCheck)+b];
               if(laneChar!=0xFF){
               gap_b=((float)initShift-b)*cellSize;//m
               v_b=laneChar;//laneChar is in 3*ms (to save space in array)
               found=true;
               break;
               }
               }
               //printf("3initShift %u numOfCells %u\n",initShift,numOfCells);
               if(found==false){
               gap_b=1000.0f;
               }

               }//

               __device__ void calculateLaneCarShouldBe(
               ushort curEdgeLane,
               ushort nextEdge,
               LC::intersectionData *intersections,
               ushort edgeNextInters,
               ushort edgeNumLanes,
               ushort &initOKLanes,
               ushort &endOKLanes){
               initOKLanes=0;
               endOKLanes=edgeNumLanes;
               bool currentEdgeFound=false;
               bool exitFound=false;
               ushort numExitToTake=0;
               ushort numExists=0;
               for(int eN=intersections[edgeNextInters].totalInOutEdges-1;eN>=0;eN--){//clockwise
               uint procEdge=intersections[edgeNextInters].edge[eN];
               if((procEdge&0xFFFF)==curEdgeLane){//current edge
               //if(DEBUG_TRAFFIC==1)printf("CE procEdge %05x\n",procEdge);
               currentEdgeFound=true;
               if(exitFound==false)
               numExitToTake=0;
               continue;
               }


               if((procEdge&0x010000)==0x0){//out edge
               //if(DEBUG_TRAFFIC==1)printf("   procEdge %05x\n",procEdge);
               numExists++;
               if(currentEdgeFound==true){
               numExitToTake++;
               }
               if(currentEdgeFound==false&&exitFound==false){
               numExitToTake++;
               }
               }
               if((procEdge&0xFFFF)==nextEdge){
               exitFound=true;
               currentEdgeFound=false;
               //if(DEBUG_TRAFFIC==1)printf("NE procEdge %05x\n",procEdge);
               }
               }
               //if(DEBUG_TRAFFIC==1)printf("Num extis %u Num exit to take %u%\n",numExists,numExitToTake);
               if(edgeNumLanes==0){
               return;//printf("ERRRROR\n");
               }
               switch(edgeNumLanes){
               /// ONE LANE
               case 1:
               initOKLanes=0;
               endOKLanes=1;
               break;
               /// TWO LANE
               case 2:
               switch(numExists){
               case 1:
               case 2://all okay
               initOKLanes=0;
               endOKLanes=2;
               break;
               case 3:
               if(numExitToTake>2){//left
               initOKLanes=0;
               endOKLanes=1;
               break;
               }
               initOKLanes=1;
               endOKLanes=2;
               break;
               default:
               if(numExitToTake>=numExists-1){
               initOKLanes=0;
               endOKLanes=1;
               break;
               }
               initOKLanes=1;
               endOKLanes=2;
               break;
               }
               break;
               /// THREE LANE
               case 3:
               switch(numExists){
               case 1:
               case 2://all okay
               initOKLanes=0;
               endOKLanes=3;
               break;
               case 3:
               if(numExitToTake>2){//left
               initOKLanes=0;
               endOKLanes=1;
               break;
               }
               initOKLanes=1;
               endOKLanes=3;
               break;
               default:
               if(numExitToTake>=numExists-1){
               initOKLanes=0;
               endOKLanes=1;
               break;
               }
               initOKLanes=1;
               endOKLanes=2;
               break;
               }
               break;
               case 4:
               switch(numExists){
               case 1:
               case 2://all okay
               initOKLanes=0;
               endOKLanes=4;
               break;
               case 3:
               if(numExitToTake==1){//right
               initOKLanes=3;
               endOKLanes=4;
               }
               if(numExitToTake>3){//left
               initOKLanes=0;
               endOKLanes=1;
               break;
               }
               initOKLanes=1;
               endOKLanes=4;
               break;
               default:
               if(numExitToTake==1){//right
               initOKLanes=edgeNumLanes-1;
               endOKLanes=edgeNumLanes;
               }
               if(numExitToTake>=numExists-2){
               initOKLanes=0;
               endOKLanes=2;
               break;
               }
               initOKLanes=1;//also lane 2
               endOKLanes=edgeNumLanes;
               }
               break;
               default:
               switch(numExists){
               case 1:
               case 2://all okay
               initOKLanes=0;
               endOKLanes=edgeNumLanes;
               break;
               case 3:
               if(numExitToTake==1){//right
               initOKLanes=edgeNumLanes-1;
               endOKLanes=edgeNumLanes;
               }
               if(numExitToTake>edgeNumLanes-2){//left
               initOKLanes=0;
               endOKLanes=2;
               break;
               }
               initOKLanes=1;
               endOKLanes=edgeNumLanes;
               break;
               default:
               if(numExitToTake<2){//right
               initOKLanes=edgeNumLanes-2;
               endOKLanes=edgeNumLanes;
               }
               if(numExitToTake>=numExists-2){
               initOKLanes=0;
               endOKLanes=2;
               break;
               }
               initOKLanes=1;//also lane 2
               endOKLanes=edgeNumLanes-1;
               }
               break;

               }
               }//

               __device__ int cuda_qrand(){
               return 10;
               }

               // Kernel that executes on the CUDA device
               __global__ void kernel_trafficSimulation(
               int numPeople,
               float currentTime,
               float cellSize,
               float deltaTime,
               LC::CUDATrafficPerson *trafficPersonVec,
               //ushort *nextEdgeM,
               LC::edgeData* edgesData,
               uchar *laneMap,
               LC::intersectionData *intersections,
               uchar *trafficLights,
               uint mapToReadShift,
               uint mapToWriteShift,
               ushort maxWidth)
               {
               bool DEBUG_TRAFFIC=0;
               int p = blockIdx.x * blockDim.x + threadIdx.x;
               //printf("p %d Numpe %d\n",p,numPeople);
               if(p<numPeople){//CUDA check (inside margins)
               ///
               ///////////////////////////////
               //2.0. check if finished
               if(trafficPersonVec[p].active==2){
               return;
               }
               ///////////////////////////////
               //2.1. check if person should still wait or should start
               if(trafficPersonVec[p].active==0){

               //printf("  1. Person: %d active==0\n",p);
               if(trafficPersonVec[p].time_departure>currentTime){//wait
               //1.1 just continue waiting
               //printf("   1.1 Person: %d wait\n",p);
               return;
               }else{//start
               //1.2 find first edge
               trafficPersonVec[p].currPathEdge=0;
               ushort firstEdge=trafficPersonVec[p].personPath[0];
               if(firstEdge==0xFFFF){
               trafficPersonVec[p].active=2;
               //printf("0xFFFF\n");
               return;
               }


               //1.3 update person edgeData
               //if(DEBUG_TRAFFIC==1)printf("   1.3 Person: %d put in first edge %u\n",p,firstEdge);
               //printf("edgesData %d\n",edgesData);

               // COPY DATA FROM EDGE TO PERSON
               trafficPersonVec[p].edgeNumLanes=edgesData[firstEdge].numLines;
               trafficPersonVec[p].edgeNextInters=edgesData[firstEdge].nextInters;

               trafficPersonVec[p].length=edgesData[firstEdge].length;
               trafficPersonVec[p].maxSpeedMperSec=edgesData[firstEdge].maxSpeedMperSec;
               //printf("edgesData %.10f\n",edgesData[firstEdge].maxSpeedCellsPerDeltaTime);
               //1.4 try to place it in middle of edge
               ushort numOfCells=ceil(trafficPersonVec[p].length/cellSize);
               ushort initShift=(ushort)(0.5f*numOfCells);//number of cells it should be placed (half of road)

               uchar laneChar;
               bool placed=false;

               ushort numCellsEmptyToBePlaced=s_0/cellSize;
               ushort countEmptyCells=0;
               for(ushort b=initShift;(b<numOfCells)&&(placed==false);b++){
               //for(int lN=trafficPersonVec[p].edgeNumLanes-1;lN>=0;lN--){
               //ushort lN=0;//just first LANE !!!!!!!
               ushort lN=trafficPersonVec[p].edgeNumLanes-1;//just right LANE !!!!!!!
               laneChar=laneMap[mapToReadShift+maxWidth*(firstEdge+lN)+b];//get byte of edge (proper line)
               if(laneChar!=0xFF){
               countEmptyCells=0;
               continue;
               }
               countEmptyCells++;// ensure there is enough room to place the car
               if(countEmptyCells<numCellsEmptyToBePlaced){
               continue;
               }
               trafficPersonVec[p].numOfLaneInEdge=lN;
               trafficPersonVec[p].posInLaneM=b*cellSize;//m
               uchar vInMpS=(uchar)(trafficPersonVec[p].v*3);//speed in m/s *3 (to keep more precision
               laneMap[mapToWriteShift+maxWidth*(firstEdge+lN)+b]=vInMpS;
               placed=true;
               //printf("Placed\n");
               break;
               //}
               }
               if(placed==false){//not posible to start now
               return;
               }
               trafficPersonVec[p].v=0;//trafficPersonVec[p].maxSpeedCellsPerDeltaTime;//(20000.0f*deltaTime)/cellSize;//20km/h-->cell/delta time
               trafficPersonVec[p].LC_stateofLaneChanging=0;
               //1.5 active car

               trafficPersonVec[p].active=1;
               trafficPersonVec[p].num_steps=1;
               trafficPersonVec[p].gas=0;
               //trafficPersonVec[p].nextPathEdge++;//incremet so it continues in next edge
               // set up next edge info
               ushort nextEdge=trafficPersonVec[p].personPath[1];
               //trafficPersonVec[p].nextEdge=nextEdge;
               if(nextEdge!=0xFFFF){
               trafficPersonVec[p].nextEdgemaxSpeedMperSec=edgesData[nextEdge].maxSpeedMperSec;
               trafficPersonVec[p].nextEdgeNumLanes=edgesData[nextEdge].numLines;
               trafficPersonVec[p].nextEdgeNextInters=edgesData[nextEdge].nextInters;
               trafficPersonVec[p].nextEdgeLength=edgesData[nextEdge].length;
               //trafficPersonVec[p].nextPathEdge++;
               trafficPersonVec[p].LC_initOKLanes=0xFF;
               trafficPersonVec[p].LC_endOKLanes=0xFF;
               }
               return;
               }
               }

               ///////////////////////////////
               //2. it is moving
               trafficPersonVec[p].num_steps++;
               //2.1 try to move
               float numMToMove;
               bool getToNextEdge=false;
               bool nextVehicleIsATrafficLight=false;
               ushort currentEdge=trafficPersonVec[p].personPath[trafficPersonVec[p].currPathEdge];
               ushort nextEdge=trafficPersonVec[p].personPath[trafficPersonVec[p].currPathEdge+1];
               //if(trafficPersonVec[p].posInLaneM<trafficPersonVec[p].length){
               // www.vwi.tu-dresden.de/~treiber/MicroApplet/IDM.html
               // IDM
               float thirdTerm=0;
               ///////////////////////////////////////////////////
               // 2.1.1 Find front car
               //int numCellsCheck=fmax<float>(15.0f,trafficPersonVec[p].v*deltaTime*2);//15 or double of the speed*time
               // SAME LINE
               bool found=false;
               float s;
               float delta_v;
               uchar laneChar;
               ushort byteInLine=(ushort)floor(trafficPersonVec[p].posInLaneM/cellSize);
               ushort numOfCells=ceil(trafficPersonVec[p].length/cellSize);
               for(ushort b=byteInLine+2;(b<numOfCells)&&(found==false);b++){
               laneChar=laneMap[mapToReadShift+maxWidth*(trafficPersonVec[p].personPath[trafficPersonVec[p].currPathEdge]+trafficPersonVec[p].numOfLaneInEdge)+b];
               if(laneChar!=0xFF){
               s=((float)(b-byteInLine)*cellSize);//m
               delta_v=trafficPersonVec[p].v-(laneChar/3.0f);//laneChar is in 3*ms (to save space in array)
               found=true;
               //printf("\nFOUND Car Same Lane s %f delta_v %f!!!!\n",s,delta_v);
               break;
               }
               }
               // TRAFFIC LIGHT
               if(found==false){//check if traffic light is red
               if(trafficLights[currentEdge]==0x00){//red
               s=((float)(numOfCells-byteInLine)*cellSize);//m
               delta_v=trafficPersonVec[p].v-0;//it should be treated as an obstacle
               nextVehicleIsATrafficLight=true;
               //printf("\nFOUND TL\n",s,delta_v);
               found=true;
               }
               }
               // NEXT LINE
               if(found==false){//check if in next line
               if((nextEdge!=0xFFFF)&&(trafficPersonVec[p].edgeNextInters!=trafficPersonVec[p].end_intersection)){// we haven't arrived to destination (check next line)

               ushort nextEdgeLaneToBe=trafficPersonVec[p].numOfLaneInEdge;//same lane
               //printf("trafficPersonVec[p].numOfLaneInEdge %u\n",trafficPersonVec[p].numOfLaneInEdge);
               if(nextEdgeLaneToBe>=trafficPersonVec[p].nextEdgeNumLanes){
               nextEdgeLaneToBe=trafficPersonVec[p].nextEdgeNumLanes-1;//change line if there are less roads
               }
               //printf("2trafficPersonVec[p].numOfLaneInEdge %u\n",trafficPersonVec[p].numOfLaneInEdge);
               ushort numOfCells=ceil(trafficPersonVec[p].nextEdgeLength/cellSize);
               for(ushort b=0;(b<numOfCells)&&(found==false);b++){
               laneChar=laneMap[mapToReadShift+maxWidth*(nextEdge+nextEdgeLaneToBe)+b];
               if(laneChar!=0xFF){
               s=((float)(b)*cellSize);//m
               delta_v=trafficPersonVec[p].v-(laneChar/3.0f);//laneChar is in 3*ms (to save space in array)
               found=true;
               //printf("\FOUND Car Next Lane s %f delta_v %f!!!!\n",s,delta_v);
               //printf("\FOUND Car Next Lane next Edge %u NumLanes %u Be in Lane %u!!!!\n",trafficPersonVec[p].nextEdge,trafficPersonVec[p].nextEdgeNumLanes,nextEdgeLaneToBe);
               break;
               }
               }
               }

               }
               float s_star;
               if(found==true){//car in front and slower than us
               // 2.1.2 calculate dv_dt
               s_star=s_0+fmax(0.0f,(trafficPersonVec[p].v*trafficPersonVec[p].T+(trafficPersonVec[p].v*delta_v)/(2*std::sqrt(trafficPersonVec[p].a*trafficPersonVec[p].b))));
               thirdTerm=std::pow(((s_star)/(s)),2);
               //printf(">FOUND s_star %f thirdTerm %f!!!!\n",s_star,thirdTerm);
               }

               float dv_dt= trafficPersonVec[p].a*(1.0f-std::pow((trafficPersonVec[p].v/trafficPersonVec[p].maxSpeedMperSec),4)-thirdTerm);

               // 2.1.3 update values
               numMToMove=fmax(0.0f,trafficPersonVec[p].v*deltaTime+0.5f*(dv_dt)*deltaTime*deltaTime);


               //printf("v %.10f v d %.10f\n",trafficPersonVec[p].v,trafficPersonVec[p].v+((dv_dt/(deltaTime)/deltaTime)));
               trafficPersonVec[p].v+=dv_dt*deltaTime;
               if(trafficPersonVec[p].v<0){
               //printf("p %d v %f v0 %f a %f dv_dt %f s %f s_star %f MOVE %f\n",p,trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec,trafficPersonVec[p].a,dv_dt,s,s_star,numMToMove);
               trafficPersonVec[p].v=0;
               }
               /////
               //CO2
               //if(trafficPersonVec[p].v>0)
               {
               float speedMph=trafficPersonVec[p].v*2.2369362920544;//mps to mph
               float gasStep=-0.064+0.0056*speedMph+0.00026*(speedMph-50.0f)*(speedMph-50.0f);
               if(gasStep>0){
               gasStep*=deltaTime;
               trafficPersonVec[p].gas+=gasStep;
               }
               }
               //trafficPersonVec[p].gas+=numMToMove/1000.0f;
               //////////////////////////////////////////////

               if(trafficPersonVec[p].v==0){//if not moving not do anything else
               ushort posInLineCells=(ushort)(trafficPersonVec[p].posInLaneM/cellSize);
               laneMap[mapToWriteShift+maxWidth*(currentEdge+trafficPersonVec[p].numOfLaneInEdge)+posInLineCells]=0;
               return;
               }
               //////////

               ///////////////////////////////
               // COLOR
               ////////////////////////////////


               //numCellsToMove=trafficPersonVec[p].v;
               trafficPersonVec[p].posInLaneM=trafficPersonVec[p].posInLaneM+numMToMove;
               if(trafficPersonVec[p].posInLaneM>trafficPersonVec[p].length){//research intersection
               numMToMove=trafficPersonVec[p].posInLaneM-trafficPersonVec[p].length;
               getToNextEdge=true;
               }else{//does not research next intersection
               ////////////////////////////////////////////////////////
               // LANE CHANGING (happens when we are not reached the intersection)
               if(trafficPersonVec[p].v>3.0f&&//at least 10km/h to try to change lane
               trafficPersonVec[p].num_steps%10==0//just check every (10 steps) 5 seconds
               ){
               //next thing is not a traffic light
               // skip if there is one lane (avoid to do this)
               // skip if it is the last edge
               if(nextVehicleIsATrafficLight==false&&trafficPersonVec[p].edgeNumLanes>1&&nextEdge!=0xFFFF){

               ////////////////////////////////////////////////////
               // LC 1 update lane changing status
               if(trafficPersonVec[p].LC_stateofLaneChanging==0){
               // 2.2-exp((x-1)^2)
               float x=trafficPersonVec[p].posInLaneM/trafficPersonVec[p].length;
               if(x>0.4f){//just after 40% of the road
               float probabiltyMandatoryState=2.2-exp((x-1)*(x-1));
               {
               trafficPersonVec[p].LC_stateofLaneChanging=1;
               }
               }

               }

               ////////////////////////////////////////////////////
               // LC 2 NOT MANDATORY STATE
               if(trafficPersonVec[p].LC_stateofLaneChanging==0){
               //if(p==40)printf("LC v %f v0 %f a %f\n",trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec*0.5f,dv_dt);
               // discretionary change: v slower than the current road limit and deccelerating and moving
               if((trafficPersonVec[p].v<(trafficPersonVec[p].maxSpeedMperSec*0.7f))&&(dv_dt<0)&&trafficPersonVec[p].v>3.0f){
               //printf(">>LANE CHANGE\n");
               ushort laneToCheck;//!!!!
               //printf("LC 0 %u\n",trafficPersonVec[p].numOfLaneInEdge);
               bool leftLane=trafficPersonVec[p].numOfLaneInEdge>0;//at least one lane on the left
               bool rightLane=trafficPersonVec[p].numOfLaneInEdge<trafficPersonVec[p].edgeNumLanes-1;//at least one lane
               if(leftLane==true&&rightLane==true){
               if((int(currentTime))%2==0){
               leftLane=false;
               }else{
               rightLane=false;
               }
               }
               if(leftLane==true){
               laneToCheck=trafficPersonVec[p].numOfLaneInEdge-1;
               }else{
               laneToCheck=trafficPersonVec[p].numOfLaneInEdge+1;
               }

               uchar v_a,v_b;float gap_a,gap_b;
               //printf("p %u LC 1 %u\n",p,laneToCheck);
               uchar trafficLightState=trafficLights[currentEdge];
               calculateGapsLC(cellSize,maxWidth,mapToReadShift,laneMap,trafficLightState,currentEdge+laneToCheck,trafficPersonVec[p].posInLaneM,trafficPersonVec[p].length,v_a,v_b,gap_a,gap_b);
               //printf("LC 2 %u %u %f %f\n",v_a,v_b,gap_a,gap_b);
               if(gap_a==1000.0f&&gap_b==1000.0f){//lag and lead car very far
               trafficPersonVec[p].numOfLaneInEdge=laneToCheck;// CHANGE LINE

               }else{// NOT ALONE
               float b1A=0.05f,b2A=0.15f;
               float b1B=0.15f,b2B=0.40f;
               // s_0-> critical lead gap
               float g_na_D,g_bn_D;
               bool acceptLC=true;
               if(gap_a!=1000.0f){
               g_na_D=fmax(s_0,s_0+b1A*trafficPersonVec[p].v+b2A*(trafficPersonVec[p].v-v_a*3.0f));
               if(gap_a<g_na_D)//gap smaller than critical gap
               acceptLC=false;
               }
               if(acceptLC==true&&gap_b!=1000.0f){
               g_bn_D=fmax(s_0,s_0+b1B*v_b*3.0f+b2B*(v_b*3.0f-trafficPersonVec[p].v));
               if(gap_b<g_bn_D)//gap smaller than critical gap
               acceptLC=false;
               }
               if(acceptLC==true){
               trafficPersonVec[p].numOfLaneInEdge=laneToCheck;// CHANGE LINE
               }
               }
               //printf("<<LANE CHANGE\n");
               }


               }// Discretionary

               ////////////////////////////////////////////////////
               // LC 3 *MANDATORY* STATE
               if(trafficPersonVec[p].LC_stateofLaneChanging==1){
               // LC 3.1 Calculate the correct lanes
               if(trafficPersonVec[p].LC_endOKLanes==0xFF){
               calculateLaneCarShouldBe(currentEdge,nextEdge,intersections,trafficPersonVec[p].edgeNextInters,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
               //printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
               if(trafficPersonVec[p].LC_initOKLanes==0&&trafficPersonVec[p].LC_endOKLanes==0)
               return;
               }


               //printf(">>LANE CHANGE\n");
               ushort laneToCheck;//!!!!
               //printf("LC 0 %u\n",trafficPersonVec[p].numOfLaneInEdge);
               bool leftLane=false,rightLane=false;
               // LC 3.2 CORRECT LANES--> DICRETIONARY LC WITHIN
               if(trafficPersonVec[p].numOfLaneInEdge>=trafficPersonVec[p].LC_initOKLanes&&trafficPersonVec[p].numOfLaneInEdge<trafficPersonVec[p].LC_endOKLanes){
               // for discretionary it should be under some circustances
               if((trafficPersonVec[p].v<(trafficPersonVec[p].maxSpeedMperSec*0.7f))&&(dv_dt<0)&&trafficPersonVec[p].v>3.0f){
               leftLane=
               (trafficPersonVec[p].numOfLaneInEdge>0) &&//at least one lane on the left
               (trafficPersonVec[p].numOfLaneInEdge-1>=trafficPersonVec[p].LC_initOKLanes)&&
               (trafficPersonVec[p].numOfLaneInEdge-1<trafficPersonVec[p].LC_endOKLanes);
               rightLane=
               (trafficPersonVec[p].numOfLaneInEdge<trafficPersonVec[p].edgeNumLanes-1)&&//at least one lane
               (trafficPersonVec[p].numOfLaneInEdge+1>=trafficPersonVec[p].LC_initOKLanes)&&
               (trafficPersonVec[p].numOfLaneInEdge+1<trafficPersonVec[p].LC_endOKLanes);
               //printf("D\n");
               }
               }
               // LC 3.3 INCORRECT LANES--> MANDATORY LC
               else{
               //printf("num lanes %u min %u max %u\n",trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
               //printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);

               if(trafficPersonVec[p].numOfLaneInEdge<trafficPersonVec[p].LC_initOKLanes){
               rightLane=true;
               }else{
               leftLane=true;
               }
               if(rightLane==true&&trafficPersonVec[p].numOfLaneInEdge+1>=trafficPersonVec[p].edgeNumLanes){

               }
               if(leftLane==true&&trafficPersonVec[p].numOfLaneInEdge==0){

               return;
               }
               //printf("M L %d R %d nL %u\n",leftLane,rightLane,trafficPersonVec[p].numOfLaneInEdge);
               }
               if(leftLane==true||rightLane==true){

               // choose lane (if necessary)
               if(leftLane==true&&rightLane==true){
               if((int(currentTime))%2==0){
               leftLane=false;
               }else{
               rightLane=false;
               }
               }
               if(leftLane==true){
               laneToCheck=trafficPersonVec[p].numOfLaneInEdge-1;
               }else{
               laneToCheck=trafficPersonVec[p].numOfLaneInEdge+1;
               }
               if(laneToCheck>=trafficPersonVec[p].edgeNumLanes){

               }
               uchar v_a,v_b;float gap_a,gap_b;
               //printf("p %u LC 1 %u\n",p,laneToCheck);
               uchar trafficLightState=trafficLights[currentEdge];
               calculateGapsLC(cellSize,maxWidth,mapToReadShift,laneMap,trafficLightState,currentEdge+laneToCheck,trafficPersonVec[p].posInLaneM,trafficPersonVec[p].length,v_a,v_b,gap_a,gap_b);
               //printf("LC 2 %u %u %f %f\n",v_a,v_b,gap_a,gap_b);
               if(gap_a==1000.0f&&gap_b==1000.0f){//lag and lead car very far
               trafficPersonVec[p].numOfLaneInEdge=laneToCheck;// CHANGE LINE

               }else{// NOT ALONE
               float b1A=0.05f,b2A=0.15f;
               float b1B=0.15f,b2B=0.40f;
               float gamma=0.000025;
               // s_0-> critical lead gap
               float distEnd=trafficPersonVec[p].length-trafficPersonVec[p].posInLaneM;
               float expTerm=(1-exp(-gamma*distEnd*distEnd));

               float g_na_M,g_bn_M;
               bool acceptLC=true;
               if(gap_a!=1000.0f){
               g_na_M=fmax(s_0,s_0+(b1A*trafficPersonVec[p].v+b2A*(trafficPersonVec[p].v-v_a*3.0f)));
               if(gap_a<g_na_M)//gap smaller than critical gap
               acceptLC=false;
               }
               if(acceptLC==true&&gap_b!=1000.0f){
               g_bn_M=fmax(s_0,s_0+(b1B*v_b*3.0f+b2B*(v_b*3.0f-trafficPersonVec[p].v)));
               if(gap_b<g_bn_M)//gap smaller than critical gap
               acceptLC=false;
               }
               if(acceptLC==true){
               trafficPersonVec[p].numOfLaneInEdge=laneToCheck;// CHANGE LINE
               }
               }


               }

               }// Mandatory

               }//at least two lanes and not stopped by traffic light

               }
               ///////////////////////////////////////////////////////

               uchar vInMpS=(uchar)(trafficPersonVec[p].v*3);//speed in m/s to fit in uchar
               ushort posInLineCells=(ushort)(trafficPersonVec[p].posInLaneM/cellSize);
               laneMap[mapToWriteShift+maxWidth*(currentEdge+trafficPersonVec[p].numOfLaneInEdge)+posInLineCells]=vInMpS;
               //printf("2<<LANE CHANGE\n");
               return;
               }
               //}
               //2.2 close to intersection

               //2.2 check if change intersection
               //!!!ALWAYS CHANGE
               //2.2.1 find next edge
               if(nextEdge==0xFFFF){//if(curr_intersection==end_intersection){

               trafficPersonVec[p].active=2;//finished
               return;
               }
               //if(trafficPersonVec[p].nextPathEdge>=nextEdgeM.size())printf("AAAAAAAAAAAAAAAAA\n");
               /////////////
               // update edge
               //trafficPersonVec[p].curEdgeLane=trafficPersonVec[p].nextEdge;
               trafficPersonVec[p].currPathEdge++;
               trafficPersonVec[p].maxSpeedMperSec=trafficPersonVec[p].nextEdgemaxSpeedMperSec;
               trafficPersonVec[p].edgeNumLanes=trafficPersonVec[p].nextEdgeNumLanes;
               trafficPersonVec[p].edgeNextInters=trafficPersonVec[p].nextEdgeNextInters;
               trafficPersonVec[p].length=trafficPersonVec[p].nextEdgeLength;
               trafficPersonVec[p].posInLaneM=numMToMove;
               if(trafficPersonVec[p].numOfLaneInEdge>=trafficPersonVec[p].edgeNumLanes){
               trafficPersonVec[p].numOfLaneInEdge=trafficPersonVec[p].edgeNumLanes-1;//change line if there are less roads
               }

               ////////////
               // update next edge
               ushort nextNEdge=trafficPersonVec[p].personPath[trafficPersonVec[p].currPathEdge+1];
               //trafficPersonVec[p].nextEdge=nextEdge;
               if(nextNEdge!=0xFFFF){
               //trafficPersonVec[p].nextPathEdge++;
               trafficPersonVec[p].LC_initOKLanes=0xFF;
               trafficPersonVec[p].LC_endOKLanes=0xFF;

               //2.2.3 update person edgeData
               //trafficPersonVec[p].nextEdge=nextEdge;
               trafficPersonVec[p].nextEdgemaxSpeedMperSec=edgesData[nextNEdge].maxSpeedMperSec;
               trafficPersonVec[p].nextEdgeNumLanes=edgesData[nextNEdge].numLines;
               trafficPersonVec[p].nextEdgeNextInters=edgesData[nextNEdge].nextInters;
               trafficPersonVec[p].nextEdgeLength=edgesData[nextNEdge].length;
               }

               trafficPersonVec[p].LC_stateofLaneChanging=0;
               uchar vInMpS=(uchar)(trafficPersonVec[p].v*3);//speed in m/s to fit in uchar
               ushort posInLineCells=(ushort)(trafficPersonVec[p].posInLaneM/cellSize);
               laneMap[mapToWriteShift+maxWidth*(nextEdge+trafficPersonVec[p].numOfLaneInEdge)+posInLineCells]=vInMpS;

               ///
               }

               }//

               __global__ void kernel_intersectionSimulation(uint numIntersections,float currentTime,LC::intersectionData *intersections,uchar *trafficLights) {
               int i = blockIdx.x * blockDim.x + threadIdx.x;
               if(i<numIntersections){//CUDA check (inside margins)

               //////////////////////////////////////////////////////
               const float deltaEvent=20.0f;
               //if(i==0)printf("i %d\n",i);
               if(currentTime>intersections[i].nextEvent&&intersections[i].totalInOutEdges>0){


               uint edgeOT=intersections[i].edge[intersections[i].state];
               uchar numLinesO=edgeOT>>24;
               ushort edgeONum=edgeOT&0xFFFF;
               // red old traffic lights
               for(int nL=0;nL<numLinesO;nL++){
               trafficLights[edgeONum+nL]=0x00;//red old traffic light
               }
               for(int iN=0;iN<=intersections[i].totalInOutEdges;iN++){//to give a round
               intersections[i].state=(intersections[i].state+1)%intersections[i].totalInOutEdges;//next light
               if((intersections[i].edge[intersections[i].state]&0x010000)==0x010000){
               // green new traffic lights
               uint edgeIT=intersections[i].edge[intersections[i].state];
               ushort edgeINum=edgeIT&0xFFFF;//get edgeI
               uchar numLinesI=edgeIT>>24;
               for(int nL=0;nL<numLinesI;nL++){
               trafficLights[edgeINum+nL]=0xFF;
               }
               break;
               }
               }//green new traffic light
               //printf("i %d CHANGE state %u of %d (Old edge %u New Edge %u)\n",i,intersections[i].state,intersections[i].totalInOutEdges,edgeO,edgeI);
               ////
               intersections[i].nextEvent=currentTime+deltaEvent;
               }


               //////////////////////////////////////////////////////
               }

               }//


               void simulateTrafficCUDA(float currentTime,uint numPeople,uint numIntersections){

               ////////////////////////////////////////////////////////////
               // 1. CHANGE MAP: set map to use and clean the other
               if(readFirstMap==true){
               mapToReadShift=0;
               mapToWriteShift=halfLaneMap;
               hipMemset(&laneMap_d[halfLaneMap],-1,halfLaneMap*sizeof (unsigned char));//clean second half
               }else{
               mapToReadShift=halfLaneMap;
               mapToWriteShift=0;
               hipMemset(&laneMap_d[0],-1,halfLaneMap*sizeof (unsigned char));//clean first half
               }
               readFirstMap=!readFirstMap;//next iteration invert use


               kernel_intersectionSimulation <<< numIntersections,1 >>> (numIntersections, currentTime,intersections_d,trafficLights_d);
               kernel_trafficSimulation <<< ceil(numPeople/1024.0f),1024 >>> (numPeople,currentTime,cellSize,deltaTime,trafficPersonVec_d,edgesData_d,laneMap_d,intersections_d,trafficLights_d, mapToReadShift,mapToWriteShift,maxWidth);// n_blocks, block_size >>> (a_d, N);
               }//
               */
